#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <random>
#include <cmath>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand/hiprand.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::stringstream err;
        err << "GPUassert -- " << file << "(" << line << "): " << hipGetErrorString(code) << std::endl;
        std::cout << err.str(); // Needed on Windows for some reason
        throw std::runtime_error(err.str());
    }
} 

// The GPU kernel to check if the randomly generated (x,y) pairs fall in the unit circle in the first quadrant or not.
// This is assuming that the total number of (x,y) pairs is an exact multiple of 1024 so that all threads in every
// thread block will be doing work and no out of bounds memory errors occur.
__global__ void checkIn(float *x, float *y, int *counts) {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    
    __shared__ int result[1024];
    
    float r = sqrtf(x[tid]*x[tid] + y[tid]*y[tid]);
    result[threadIdx.x] = int(2 - r);
    __syncthreads();
    
    if (threadIdx.x < 31) {
        for (int i = 0; i < 32; ++i) {
            result[threadIdx.x] += result[(threadIdx.x+1)*32 + i];
        }
    }
    __syncthreads();
    
    if (threadIdx.x == 0) {
        for (int i = 1; i < 32; ++i) {
            result[threadIdx.x] += result[i];
        }
        counts[blockIdx.x] = result[threadIdx.x];
    }
}

// This GPU kernel takes the results from checkIn, does a reduction of the results from each block and calculates a
// Monte Carlo estimate of pi which is stored in the array pis.
__global__ void getPi(int *counts, double *pis, int iter) {
    for (int i = 0; i < 32; ++i) {
        counts[threadIdx.x] += counts[(threadIdx.x + 1)*32 + i];
    }
    __syncthreads();
    
    if (threadIdx.x == 0) {
        for (int i = 1; i < 32; ++i) {
            counts[0] += counts[i];
        }
        pis[iter] = 4.0*double(counts[0])/1048576.0;
    }
}

int main() {
    size_t N_blocks = 1024;
    size_t N_threads = 1024;
    int iter = 1000;
    size_t N_pairs = N_blocks*N_threads;
    int *d_counts;
    float *d_x, *d_y;
    std::vector<double> pis(iter);
    std::vector<float> x(N_pairs), y(N_pairs);
    std::vector<int> counts(N_blocks);
    double *d_pis;
    hiprandGenerator_t gen;
    std::random_device seeder;
    
    hipMalloc((void**)&d_counts, N_blocks*sizeof(int));
    hipMalloc((void**)&d_x, N_pairs*sizeof(float));
    hipMalloc((void**)&d_y, N_pairs*sizeof(float));
    hipMalloc((void**)&d_pis, iter*sizeof(double));
    
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(gen, seeder());
    
    double t_1 = omp_get_wtime();
    for (int i = 0; i < iter; ++i) {
        hiprandGenerateUniform(gen, d_x, N_pairs);
        hiprandGenerateUniform(gen, d_y, N_pairs);
        
        checkIn<<<N_blocks,N_threads>>>(d_x, d_y, d_counts);
        
        getPi<<<1,31>>>(d_counts, d_pis, i);
    }
    double t_2 = omp_get_wtime();
    std::cout.precision(15);
    std::cout << "Time on GPU: " << t_2 - t_1 << " s\n";
    
    hipMemcpy(pis.data(), d_pis, iter*sizeof(double), hipMemcpyDeviceToHost);
    
    std::ofstream fout("pisCUDA.dat");
    fout.precision(15);
    for (int i = 0; i < iter; ++i) {
        //std::cout << pis[i] << "\n";
        fout << pis[i] << "\n";
    }
    fout.close();
    
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_counts);
    hipFree(d_pis);
    hiprandDestroyGenerator(gen);
    
    return 0;
}
